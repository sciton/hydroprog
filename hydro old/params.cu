#include "params.h"
//#include "definitions.h"
#include <math.h>
#include <stdlib.h>

float *x;
float *y;
float *Vx;
float *Vy;
float *ax;
float *ay;


extern "C" void InitData()
{
    int numBytes = NumP*sizeof(float);
  gpuErrchk(hipHostAlloc((void**)&x,numBytes,
  hipHostMallocWriteCombined | hipHostMallocMapped ));
  gpuErrchk(hipHostAlloc((void**)&y,numBytes,
  hipHostMallocWriteCombined | hipHostMallocMapped ));
  gpuErrchk(hipHostAlloc((void**)&Vx,numBytes,
  hipHostMallocWriteCombined | hipHostMallocMapped ));
  gpuErrchk(hipHostAlloc((void**)&Vy,numBytes,
  hipHostMallocWriteCombined | hipHostMallocMapped ));
  gpuErrchk(hipHostAlloc((void**)&ax,numBytes,
  hipHostMallocWriteCombined | hipHostMallocMapped ));
  gpuErrchk(hipHostAlloc((void**)&ay,numBytes,
  hipHostMallocWriteCombined | hipHostMallocMapped ));

  int i;
  for (i=0; i<NumP; i++)
  {
    do
    {
       x[i] = lbound + (rbound - lbound)*rand()/RAND_MAX;
       y[i] = bbound + (tbound - bbound)*rand()/RAND_MAX;
    } while(( x[i]- circX)*( x[i]- circX)+
  ( y[i]-circY)*( y[i]-circY)<circR*circR);
     Vx[i] = initVx;
     Vy[i] = initVy;
     ax[i] = initax;
     ay[i] = initay;
  }
}
