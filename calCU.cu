#include "hip/hip_runtime.h"
#include "params.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>

__global__
void accelerate(
  float *devx,
  float *devy,
  float *devVx,
  float *devVy,
  float *devax,
  float *devay)
  {
    int i = threadIdx.x+blockIdx.x*blockDim.x;
    int j = 0;
    float dist;
    devax[i] = initax;
    devay[i] = initay;
    for (j = 0; j<NumP; j++)
      if (i!=j)
      {
        dist = (devx[i]-devx[j])*(devx[i]-devx[j])
        +(devy[i]-devy[j])*(devy[i]-devy[j]);
        if (dist<effrad)
        {
          devax[i] = devax[i]+pressure*(devx[i]-devx[j])/dist;
          devay[i] = devay[i]+pressure*(devy[i]-devy[j])/dist;
        }
      }
  }

__global__
void newcoord(
  float *devx,
  float *devy,
  float *devVx,
  float *devVy,
  float *devax,
  float *devay)
  {
    float r,a,b,rx,ry;

    int i = threadIdx.x+blockIdx.x*blockDim.x;

    devx[i] = devx[i]+devVx[i]*Timestep+
                  devax[i]*Timestep*Timestep/2.0;
    devy[i] = devy[i]+devVy[i]*Timestep+
                  devay[i]*Timestep*Timestep/2.0;
    devVx[i] = devVx[i] + devax[i]*Timestep;
    devVy[i] = devVy[i] + devay[i]*Timestep;
    if (devy[i] < bbound)
        {
            // Vy[i] = -Vy[i];
            devy[i] = tbound;
            devVx[i] = initVx;//5.0*rand()/RAND_MAX;
            devVy[i] = initVy;
        }
      if (devy[i] > tbound)
           devVy[i] = -devVy[i];
      if ((devx[i] > rbound) || (devx[i] < lbound))
        {
          devVx[i] = -devVx[i];
        }
      if ((devx[i]-circX)*(devx[i]-circX)+
      (devy[i]-circY)*(devy[i]-circY)<circR*circR)
      {
          rx=devx[i]-circX;
          ry=devy[i]-circY;
          r = rx*rx+ry*ry;
          a = devVx[i]*ry-devVy[i]*rx;
          b = devVx[i]*rx+devVy[i]*ry;

          devVy[i] = -((a*rx+b*ry)/r);
          devVx[i] = a/ry + (rx/ry)*devVy[i];
      }
  }

extern "C" void Step()
{
  float *devx;
  float *devy;
  float *devVx;
  float *devVy;
  float *devax;
  float *devay;

  float *circx;
  float *circy;
  float *circr;

hipProfilerStart();
  int numBytes = NumP*sizeof(float);
  hipMalloc((void**)&devx, numBytes);
  hipMalloc((void**)&devy, numBytes);
  hipMalloc((void**)&devVx, numBytes);
  hipMalloc((void**)&devVy, numBytes);
  hipMalloc((void**)&devax, numBytes);
  hipMalloc((void**)&devay, numBytes);
  hipMalloc((void**)&circx, sizeof(float));
  hipMalloc((void**)&circy, sizeof(float));
  hipMalloc((void**)&circr, sizeof(float));

  hipMemcpy(devx, &x[0], numBytes, hipMemcpyHostToDevice);
  hipMemcpy(devy, &y[0], numBytes, hipMemcpyHostToDevice);
  hipMemcpy(devVx, &Vx[0], numBytes, hipMemcpyHostToDevice);
  hipMemcpy(devVy, &Vy[0], numBytes, hipMemcpyHostToDevice);
  hipMemcpy(devax, &ax[0], numBytes, hipMemcpyHostToDevice);
  hipMemcpy(devay, &ay[0], numBytes, hipMemcpyHostToDevice);
  // hipMemcpy(circx, &circX, sizeof(float), hipMemcpyHostToDevice);
  // hipMemcpy(circy, &circY, sizeof(float), hipMemcpyHostToDevice);
  // hipMemcpy(circr, &circR, sizeof(float), hipMemcpyHostToDevice);


  dim3 threads = dim3(512,1);
  dim3 blocks = dim3((int)(NumP/threads.x),1);

  accelerate<<<blocks, threads>>>
    (devx,
      devy,
      devVx,
      devVy,
      devax,
      devay);

  newcoord<<<blocks, threads>>>
  (devx,
    devy,
    devVx,
    devVy,
    devax,
    devay);

  hipMemcpy(&x[0], devx, numBytes, hipMemcpyDeviceToHost);
  hipMemcpy(&y[0], devy, numBytes, hipMemcpyDeviceToHost);
  hipMemcpy(&Vx[0], devVx, numBytes, hipMemcpyDeviceToHost);
  hipMemcpy(&Vy[0], devVy, numBytes, hipMemcpyDeviceToHost);
  hipMemcpy(&ax[0], devax, numBytes, hipMemcpyDeviceToHost);
  hipMemcpy(&ay[0], devay, numBytes, hipMemcpyDeviceToHost);

  hipFree(devx);
  hipFree(devy);
  hipFree(devVx);
  hipFree(devVy);
  hipFree(devax);
  hipFree(devay);
  hipFree(circx);
  hipFree(circy);
  hipFree(circr);
hipProfilerStop();
  // int i = 0;
  // int j = 0;
  // float r,dist;
  // float a,b,rx,ry;
  // for (i = 0; i<NumP; i++)
  // {
  //   x[i] = x[i]+Vx[i]*Timestep+
  //                 ax[i]*Timestep*Timestep/2.0;
  //   y[i] = y[i]+Vy[i]*Timestep+
  //                 ay[i]*Timestep*Timestep/2.0;
  //   Vx[i] = Vx[i] + ax[i]*Timestep;
  //   Vy[i] = Vy[i] + ay[i]*Timestep;
  //   for (j = 0; j<NumP; j++)
  //     if (i!=j)
  //     {
  //       dist = (x[i]-x[j])*(x[i]-x[j])
  //       +(y[i]-y[j])*(y[i]-y[j]);
  //       if (dist<effrad)
  //       {
  //         ax[i] = ax[i]+(x[i]-x[j])/dist;
  //         ay[i] = ay[i]+(y[i]-y[j])/dist;
  //       }
  //     }
  //     if (y[i] < bbound)
  //       {
  //           // Vy[i] = -Vy[i];
  //           y[i] = tbound;
  //           Vx[i] = initVx;//5.0*rand()/RAND_MAX;
  //           Vy[i] = initVy;
  //       }
  //     if (y[i] > tbound)
  //          Vy[i] = -Vy[i];
  //     if ((x[i] > rbound) || (x[i] < lbound))
  //       {
  //         Vx[i] = -Vx[i];
  //       }
  //     if ((x[i]-circX)*(x[i]-circX)+
  //     (y[i]-circY)*(y[i]-circY)<circR*circR)
  //     {
  //         rx=x[i]-circX;
  //         ry=y[i]-circY;
  //         r = rx*rx+ry*ry;
  //         a = Vx[i]*ry-Vy[i]*rx;
  //         b = Vx[i]*rx+Vy[i]*ry;
  //
  //         Vy[i] = -((a*rx+b*ry)/r);
  //         Vx[i] = a/ry + (rx/ry)*Vy[i];
  //     }
  // }
}
